// rUN: %clang_cc1 -std=c++11 -fsyntax-only -verify -DHOST -verify-ignore-unexpected=note %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -fcuda-is-device -verify -verify-ignore-unexpected=note %s

#include "Inputs/hip/hip_runtime.h"

__device__ int device_fn() { return 0; }

struct MemberInitializer {
  int a = device_fn();
};

struct BaseWithDeviceConstructor {
  __device__ BaseWithDeviceConstructor() { device_fn(); }
  // Also try this with a host+device constructor.
};
struct DerivedFromBaseWithDeviceConstructor : BaseWithDeviceConstructor {};

struct BaseWithDeviceDestructor {
  __device__ ~BaseWithDeviceDestructor() { device_fn(); }
};
struct DerivedFromBaseWithDeviceDestructor : BaseWithDeviceDestructor {};

void host_fn() {
  MemberInitializer mi; // expected-error {{}}
  BaseWithDeviceConstructor bc; // expected-error {{no matching constructor}}
  BaseWithDeviceDestructor bd; // expected-error {{}}
  DerivedFromBaseWithDeviceConstructor dfbc; // expected-error {{no matching constructor}}
  DerivedFromBaseWithDeviceDestructor dfbd; // expected-error {{}}
}

struct ConstructorDefaultArg {
  ConstructorDefaultArg(int = device_fn()) {}  // expected-error {{}}
};

void host_fn_with_default_arg(int = device_fn()) {}  // expected-error {{}}

struct A {
  __device__ void operator delete(void*) {}
  virtual ~A() {}  // expected-error {{}}
};
// TODO: Same tests as above, but with HD functions.

__constant__ int x;
void host_fn() {
  x = 42;  // should be disallowed; can't write to constant memory like this.
  int y = x;  // should be disallowed; can't read from constant memory like this.
}
