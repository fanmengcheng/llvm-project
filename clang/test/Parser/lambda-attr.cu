
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -verify %s
// RUN: %clang_cc1 -std=c++11 -fsyntax-only -fcuda-is-device -verify %s

// expected-no-diagnostics

__attribute__((device)) void device_attr() {
  ([]() __attribute__((device)){})();
  ([] __attribute__((device)) () {})();
  ([] __attribute__((device)) {})();

  ([&]() __attribute__((device)){})();
  ([&] __attribute__((device)) () {})();
  ([&] __attribute__((device)) {})();

  ([&](int) __attribute__((device)){})(0);
  ([&] __attribute__((device)) (int) {})(0);
}

__attribute__((host)) __attribute__((device)) void host_device_attrs() {
  ([]() __attribute__((host)) __attribute__((device)){})();
  ([] __attribute__((host)) __attribute__((device)) () {})();
  ([] __attribute__((host)) __attribute__((device)) {})();

  ([&]() __attribute__((host)) __attribute__((device)){})();
  ([&] __attribute__((host)) __attribute__((device)) () {})();
  ([&] __attribute__((host)) __attribute__((device)) {})();

  ([&](int) __attribute__((host)) __attribute__((device)){})(0);
  ([&] __attribute__((host)) __attribute__((device)) (int) {})(0);
}
